#include <cstdlib>        // For malloc, free, exit
#include <hip/hip_runtime.h> // CUDA 运行时 API
#include <cupti.h>        // CUPTI API
#include <iostream>
#include <string>
#include <vector>

#include "kernel.h" // 包含 simple_kernel 的声明

// CUPTI 错误检查宏
#define CUPTI_CALL(call)                                                       \
  do {                                                                         \
    CUptiResult _status = call;                                                \
    if (_status != CUPTI_SUCCESS) {                                            \
      const char *errstr;                                                      \
      cuptiGetResultString(_status, &errstr);                                  \
      std::cerr << "CUPTI Error: " << errstr << " in " << __FILE__ << ":"      \
                << __LINE__ << std::endl;                                      \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// CUPTI 活动记录缓冲区分配回调函数
// CUPTI 在需要新的缓冲区来存储活动记录时会调用此函数
void bufferAlloc(uint8_t **buffer, size_t *size, size_t *maxNumRecords) {
  // 为活动记录分配 1MB 的缓冲区
  *size = 1024 * 1024; // 1MB
  *buffer = (uint8_t *)malloc(*size);
  if (!*buffer) {
    std::cerr << "Failed to allocate CUPTI activity buffer." << std::endl;
    exit(EXIT_FAILURE);
  }
  *maxNumRecords = 0; // CUPTI 将根据缓冲区大小自动确定最大记录数
}

// CUPTI 活动记录缓冲区释放和处理回调函数
// CUPTI 在缓冲区满或被刷新时会调用此函数
void bufferComplete(hipCtx_t context, uint32_t streamId, uint8_t *buffer,
                    size_t size, size_t validSize) {
  CUpti_Activity *record = NULL;
  // 遍历缓冲区中的所有活动记录
  while (true) {
    CUPTI_CALL(cuptiActivityGetNextRecord(buffer, size, &record));
    // 检查记录类型是否为 KERNEL
    if (record->kind == CUPTI_ACTIVITY_KIND_KERNEL) {
      CUpti_ActivityKernel *kernel = (CUpti_ActivityKernel *)record;
      std::cout << "----------------------------------------" << std::endl;
      std::cout << "Kernel Activity Record:" << std::endl;
      std::cout << "  Name: " << kernel->name << std::endl;
      std::cout << "  Correlation ID: " << kernel->correlationId << std::endl;
      std::cout << "  Device ID: " << kernel->deviceId << std::endl;
      std::cout << "  Context ID: " << kernel->contextId << std::endl;
      std::cout << "  Stream ID: " << kernel->streamId << std::endl;
      std::cout << "  Grid: (" << kernel->gridX << ", " << kernel->gridY << ", "
                << kernel->gridZ << ")" << std::endl;
      std::cout << "  Block: (" << kernel->blockX << ", " << kernel->blockY
                << ", " << kernel->blockZ << ")" << std::endl;
      std::cout << "  Static Shared Memory: " << kernel->staticSharedMemory
                << " bytes" << std::endl;
      std::cout << "  Dynamic Shared Memory: " << kernel->dynamicSharedMemory
                << " bytes" << std::endl;
      std::cout << "  Registers Per Thread: " << kernel->registersPerThread
                << std::endl;
      std::cout << "  Start Time: " << kernel->start << " ns" << std::endl;
      std::cout << "  End Time: " << kernel->end << " ns" << std::endl;
      std::cout << "  Duration: " << (kernel->end - kernel->start) << " ns"
                << std::endl;
    }
  }

  // 检查是否有被丢弃的记录
  size_t dropped;
  CUptiResult status =
      cuptiActivityGetNumDroppedRecords(context, streamId, &dropped);
  if (status == CUPTI_SUCCESS && dropped > 0) {
    std::cerr << "Warning: Dropped " << dropped << " activity records!"
              << std::endl;
  }

  // 释放由 bufferAlloc 分配的缓冲区
  free(buffer);
}

int main() {
  std::cout << "Starting CUPTI Kernel Activity Demo..." << std::endl;

  // 1. 检查 CUDA 设备并创建上下文
  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipGetDeviceCount failed: " << hipGetErrorString(cudaStatus)
              << std::endl;
    return 1;
  }
  if (deviceCount == 0) {
    std::cerr << "No CUDA devices found." << std::endl;
    return 1;
  }
  cudaStatus = hipSetDevice(0); // 选择第一个设备
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipSetDevice failed: " << hipGetErrorString(cudaStatus)
              << std::endl;
    return 1;
  }
  std::cout << "Set CUDA device to 0." << std::endl;

  // 2. 注册 CUPTI 活动记录回调函数
  CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferAlloc, bufferComplete));

  // 3. 启用 CUPTI Kernel 活动收集
  // CUPTI_ACTIVITY_KIND_KERNEL 收集核函数执行信息
  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_KERNEL));
  std::cout << "CUPTI Kernel Activity collection enabled." << std::endl;

  // 4. 启动 CUDA 核函数以生成活动记录
  std::cout << "Launching simple_kernel 5 times..." << std::endl;
  for (int i = 0; i < 5; ++i) {
    simple_kernel<<<1, 1>>>();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
      std::cerr << "Kernel launch " << i
                << " failed: " << hipGetErrorString(cudaStatus) << std::endl;
      return 1;
    }
  }

  // 确保所有 CUDA 操作完成，以便 CUPTI 能够收集到所有记录
  cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    std::cerr << "hipDeviceSynchronize failed: "
              << hipGetErrorString(cudaStatus) << std::endl;
    return 1;
  }
  std::cout << "All kernels launched and synchronized." << std::endl;

  // 5. 禁用 CUPTI 活动收集
  CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_KERNEL));
  std::cout << "CUPTI Kernel Activity collection disabled." << std::endl;

  // 6. 刷新所有剩余的活动记录
  // 这将强制 CUPTI 调用 bufferFree 回调来处理所有未处理的记录
  std::cout << "Flushing remaining CUPTI activity records..." << std::endl;
  CUPTI_CALL(cuptiActivityFlushAll(0)); // 0 表示没有超时

  std::cout << "CUPTI Kernel Activity Demo finished successfully." << std::endl;

  return 0;
}
